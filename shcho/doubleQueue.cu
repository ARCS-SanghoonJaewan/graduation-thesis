#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <pthread.h>
#include <semaphore.h>
#include <hip/hip_runtime.h>

struct pro_arg{
	void* src;
	size_t count;
};

struct con_arg{
	void* dst;
	size_t count;
};

struct unused_element{
	size_t index;
	void* buf;
}

static size_t queue_size;

static sem_t used_empty;
static sem_t used_full;
static pthread_mutex_t used_mutex = PTHREAD_MUTEX_INITIALIZER;
static size_t used_put_index = 0;
static size_t used_get_index = 0;
static void** used;

static sem_t unused_empty;
static sem_t unused_full;
static pthread_mutex_t unused_mutex = PTHREAD_MUTEX_INITIALIZER;
static size_t unused_put_index = 0;
static size_t unused_get_index = 0;
static struct unused_element* unused;

static pthread_mutex_t pro_index_mutex = PTHREAD_MUTEX_INITIALIZER;
static size_t pro_num;
static size_t pro_index = 0;

static size_t buf_size;
static void** buf;

void used_put(void* used_one){
	used[used_put_index] = used_one;
	used_put_index = (used_put_index + 1) % queue_size;
}

void* used_get(){
	void* ret = used[used_get_index];
	used_get_index = (used_get_index + 1) % queue_size;
	return ret;
}

void unused_put(struct unused_element unused_one){
	unused[unused_put_index] = unused_one;
	unused_put_index = (unused_put_index + 1) % queue_size;
}

struct unused_element unused_get(){
	struct unused_element ret = unused[unused_get_index];
	unused_get_index = (unused_get_index + 1) % queue_size;
	return ret;
}

int constructBuf(size_t b_size, size_t p_num, size_t q_size){
	hipError_t ret;

	buf_size = b_size;
	pro_num = p_num;
	queue_size = q_size;

	if(0 != sem_init(&used_empty, 0, 0))
		return -1;
	if(0 != sem_init(&used_full, 0, queue_size))
		return -1;
	if(0 != sem_init(&unused_empty, 0, queue_size))
		return -1;
	if(0 != sem_init(&unused_full, 0, 0))
		return -1;
	
	buf = (void**)malloc(sizeof(void*) * queue_size);
	used = (void**)malloc(sizeof(void*) * queue_size);
	unused = (struct unused_element*)malloc(sizeof(struct unused_element) * queue_size);
	if(buf == NULL)
		return -1;
	if(used == NULL)
		return -1;
	if(unused == NULL)
		return -1;

	for(size_t i = 0; i < queue_size; i++){
		ret = hipHostAlloc(&buf[i], buf_size, hipHostMallocDefault | hipHostMallocWriteCombined);
		if(ret != hipSuccess){
			switch(ret){
				case hipErrorInvalidValue:
					errno = EINVAL;
					break;
				case hipErrorOutOfMemory:
					errno = ENOMEM;
					break;
				case hipErrorHostMemoryAlreadyRegistered:
					errno = EADDRINUSE;
					break;
				case hipErrorNotSupported:
					errno = ENOSYS;
					break;
			}
			return -1;
		}
	}

	for(size_t i = 0; i < queue_size; i++)
		used_put(buf[i]);

	return 0;
}

int destroyBuf(){
	hipError_t ret;

	if(0 != sem_destroy(&used_empty))
		return -1;
	if(0 != sem_destroy(&used_full))
		return -1;
	if(0 != sem_destroy(&unused_empty))
		return -1;
	if(0 != sem_destroy(&unused_full))
		return -1;

	free(used);
	free(unused);

	for(size_t i = 0; i < queue_size; i++){
		ret = hipHostFree(buf[i]);
		if(ret != hipSuccess){
			switch(ret){
				case hipErrorInvalidValue:
					errno = EINVAL;
					break;
				case hipErrorHostMemoryNotRegistered:
					errno = EFAULT;
					break;
			}
			return -1;
		}
	}

	free(buf);

	return 0;	
}

void* producer(void* arg){
	struct pro_arg* t_pro_arg = (struct pro_arg*)arg;
	void* src = t_pro_arg->src;
	size_t count = t_pro_arg->count;
	size_t target_index = 0;
	void* target_buf;
	struct unused_element target;
	bool is_end = false;

	while(true){
		pthread_mutex_lock(&pro_index_mutex);
		if(pro_index < count){
			target_index = pro_index;
			++pro_index;
			is_end = false;
		}
		else
			is_end = true;
		pthread_mutex_unlock(&pro_index_mutex);
		if(is_end == true)
			break;

		sem_wait(&used_full);
		pthread_mutex_lock(&used_mutex);
		target_buf = used_get();
		pthread_mutex_unlock(&used_mutex);
		sem_post(&used_empty);

		memcpy(target_buf, src + target_index * buf_size; buf_size);

		sem_wait(&unused_empty);
		pthread_mutex_lock(&unused_mutex);
		target.index = target_index;
		target.buf = target_buf;
		unused_put(target);
		pthread_mutex_unlock(&unused_mutex);
		sem_post(&unused_full);
	}
}

void* consumer(void* arg){
	struct con_arg* t_con_arg = (struct con_arg*)arg;
	void* dst = t_con_arg->dst;
	size_t count = t_con_arg->count;
	size_t target_index = 0;
	void* target_buf;
	struct unused_element target;

	for(size_t i = 0; i < count; i++){
		sem_wait(&unused_full);
		pthread_mutex_lock(&unused_mutex);
		target = unused_get();
		target_index = target.index;
		target_buf = target.buf;
		pthread_mutex_unlock(&unused_mutex);
		sem_post(&unused_empty);

		hipMemcpy(dst + target_index * buf_size, target_buf, buf_size, hipMemcpyHostToDevice);

		sem_wait(&used_empty);
		pthread_mutex_lock(&used_mutex);
		used_put(target_buf);
		pthread_mutex_unlock(&used_mutex);
		sem_post(&used_full);
	}
}

void cudaMemcpyFixed(void* dst, void* src, size_t count, enum hipMemcpyKind kind){
	pthread_t* pro_t = (pthread_t*)malloc(sizeof(pthread_t) * pro_num);
	pthread_t con_t;
	struct pro_arg* t_pro_arg = (struct pro_arg*)malloc(sizeof(struct pro_arg) * pro_num);
	struct con_arg t_con_arg;

	pro_index = 0;

	for(size_t i = 0; i < pro_num; i++){
		t_pro_arg[i].src = src;
		t_pro_arg[i].count = count / buf_size;
		if(pthread_create(&pro_t[i], NULL, producer, (void*)&t_pro_arg[i]) < 0){
			perror("producer thread create error : ");
			exit(0);
		}
	}

	t_con_arg.dst = dst;
	t_con_arg.count = count / buf_size;
	if(pthread_create(&con_t, NULL, consumer, (void*)&t_con_arg) < 0){
		perror("consumer thread create error : ");
		exit(0);
	}

	for(size_t i = 0; i < pro_num; i++){
		pthread_join(pro_t[i], NULL);
		printf("pro_t %lu is end\n", i);
	}
	pthread_join(con_t, NULL);
	printf("con_t is end\n");

	hipMemcpy(dst + count - count % buf_size, src + count - count % buf_size, count % buf_size, kind);
	
	free(pro_t);
	free(t_pro_arg);
}
