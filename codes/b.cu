
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <sys/syscall.h>
#include <sys/mman.h>

#define sz sizeof(int)*100
#define SYSCALL_NUMBER 326
#define PG_SIZE 64

int main(){
	int *arr;
	unsigned int i=0,j;
	arr = (int*)malloc(sz);
	memset(arr,0,sz);
	/*
	if(cudaHostRegister(arr,sz,0)!=0){
		printf("%d: pinning error!\n",getpid());
		return 0;
	}
	*/
	   if(mlock(arr,sz)!=0){
	   printf("%d: pinning error!\n",getpid());
	   return 0;
	   }
	syscall(SYSCALL_NUMBER,arr);
	for(j=0; j<50; j++)
	{
		arr[i]+=2;
		usleep(5000);
		i+=PG_SIZE;
	}
	syscall(SYSCALL_NUMBER,arr);

	for(j=0; j<50; j++)
	{
		arr[i]+=2;
		usleep(5000);
		i+=PG_SIZE;
	}
	syscall(SYSCALL_NUMBER,arr);
	return 0;
}
