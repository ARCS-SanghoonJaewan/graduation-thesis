
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define LEN 134217728

int main(void){
	char* h_arr = (char*)malloc(LEN * 4);
	char* d_arr; hipMalloc(&d_arr, LEN * 4);
	int page_size = 1 << 12;
	unsigned long i = 0;
	hipHostRegister(h_arr, LEN * 4, 0);
	//cudaHostRegister(h_arr, LEN, 0);
	//cudaHostRegister(h_arr + LEN, LEN, 0);
	//cudaHostRegister(h_arr + LEN * 2, LEN, 0);
	//cudaHostRegister(h_arr + LEN * 3, LEN, 0);
	/*
	for(i = 0; i < LEN * 4; i += page_size)
		cudaHostRegister(h_arr + page_size * i, page_size, 0);
	memset(h_arr, 1, LEN * 4);
	*/

	hipMemcpy(d_arr, h_arr, LEN * 4, hipMemcpyHostToDevice);
	//cudaMemcpy(d_arr, h_arr, LEN, cudaMemcpyHostToDevice);
	//cudaMemcpy(d_arr + LEN, h_arr + LEN, LEN, cudaMemcpyHostToDevice);
	//cudaMemcpy(d_arr + LEN * 2, h_arr + LEN * 2, LEN, cudaMemcpyHostToDevice);
	//cudaMemcpy(d_arr + LEN * 3, h_arr + LEN * 3, LEN, cudaMemcpyHostToDevice);
	/*
	for(i = 0; i < LEN * 4; i += page_size)
		cudaMemcpy(d_arr + page_size * i, h_arr + page_size * i, page_size, cudaMemcpyHostToDevice);
	*/

	hipHostUnregister(h_arr);
	//cudaHostUnregister(h_arr + LEN);
	//cudaHostUnregister(h_arr + LEN * 2);
	//cudaHostUnregister(h_arr + LEN * 3);
	/*
	for(i = 0; i < LEN * 4; i += page_size)
		cudaHostUnregister(h_arr + page_size * 3);
	*/
	free(h_arr);
	hipFree(d_arr);

	return 0;
}
