
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int main(int argc, char *argv[]){
	char *arr;
	char *d_arr;
	unsigned long long size = atoll(argv[1]);
	arr = (char*)malloc(size);
	hipMalloc(&d_arr,size);
	if(!arr){
		printf("malloc error\n");
		return 0;
	}

	hipMemcpy(d_arr,arr,size,hipMemcpyHostToDevice);

	free(arr);
	hipFree(d_arr);

	return 0;
}
