
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int main(int argc, char *argv[]){
	char *arr;
	char *d_arr;
	unsigned long long size = atoll(argv[1]);
	hipMalloc(&d_arr,size);
	hipHostAlloc(&arr,size,hipHostMallocDefault);
	if(!arr){
		printf("malloc error\n");
		return 0;
	}
	hipMemcpy(d_arr,arr,size,hipMemcpyHostToDevice);

	hipFree(d_arr);

	return 0;
}
