
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/syscall.h>

#include <sys/mman.h>

//#define SYSCALL_NUMBER 333
#define SYSCALL_NUMBER 326

int main(){
	FILE *fd = fopen("addr.txt","w+");
	int *arr;
	int *cudaArr;

	arr = (int*)malloc(sizeof(int)*100000);
	hipMalloc(&cudaArr, sizeof(int)*100000);
	int i;
	fprintf(fd,"Address of arr is: %p\n",arr);
	fprintf(fd,"Address of arr is: %p\n",&arr);
	for(i = 0; i < 100000; i++){
		arr[i] = i;
	}
	hipHostRegister(arr,sizeof(int)*100000,0);
	hipMemcpy(cudaArr, arr, sizeof(int)*100000, hipMemcpyHostToDevice);

	syscall(SYSCALL_NUMBER,arr);
	hipFree(cudaArr);

	return 0;
}
