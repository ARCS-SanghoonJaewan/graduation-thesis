
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/syscall.h>

#include <sys/mman.h>

#define handle_error(msg) \
	do { perror(msg); exit(EXIT_FAILURE); } while (0)

#define SHELL_FMT "cat /proc/%ld/maps"
#define CMD_SIZE (sizeof(SHELL_FMT) + 20)
//#define SYSCALL_NUMBER 333
#define SYSCALL_NUMBER 326

int main(){
	int *arr;
	arr = (int*)malloc(sizeof(int)*100000);
	int i;
	char cmd[CMD_SIZE];
	for(i=0; i<4096; i++){
		arr[i] = i;
	}
	printf("Address of arr is: %p\n",arr);
	syscall(SYSCALL_NUMBER, arr);
	//mlock(arr,sizeof(int)*4096);

	snprintf(cmd, CMD_SIZE, SHELL_FMT, (long)getpid());
	system(cmd);

	hipHostRegister(arr,sizeof(int)*4096,0);
	for(i=0; i<4096; i++){
		arr[i] = i+1;
	}
	printf("Address of arr is: %p\n",arr);

	printf("After mpretect()\n");
	system(cmd);

	syscall(SYSCALL_NUMBER, arr);
	return 0;
}
