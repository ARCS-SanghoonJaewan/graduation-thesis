
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <sys/syscall.h>
#include <sys/mman.h>

#define sz sizeof(char)*4096
#define PINNING_NUM 1
#define SYSCALL_NUMBER 326

int main(){
	char *arr;
	unsigned int i=0,j;
	arr = (char*)malloc(sz*8);
	memset(arr,0,sz);
	syscall(SYSCALL_NUMBER,arr);
	for(i=0; i<PINNING_NUM; i++){
		if(hipHostRegister(arr+(i*4096),sz,0)!=0){
			printf("%d: pinning error!\n",getpid());
			return 0;
		}
	}
	syscall(SYSCALL_NUMBER,arr);
	return 0;
}
